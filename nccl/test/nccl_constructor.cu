#include "../../../mpi3/nccl/communicator.hpp"
#include "../../../mpi3/main.hpp"

#include <thrust/system/cuda/memory.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <thrust/complex.h>

namespace mpi3 = boost::mpi3;

int mpi3::main(int /*argc*/, char** /*argv*/, mpi3::communicator WORLD) {
	assert(WORLD.size() == 4);  

//	hipSetDevice(world_rank); // GPU N binds to MPI rank N

	auto HEMI = WORLD < 2;

	mpi3::nccl::communicator magnesium{HEMI};
	assert(magnesium.rank() == HEMI.rank());

	using T = thrust::complex<double>;  // int64_t;
//  thust::device_vector<T, thrust::cuda::universal_allocator<T>> A(1000, world.rank());
	thrust::device_vector<T, thrust::cuda::allocator<T>> A(1000, T{1.*WORLD.rank()});

	magnesium.all_reduce_n(A.data(), A.size(), A.data());
	thrust::host_vector<T> H = A;

	std::cout<<"[WORLD rank"<< WORLD.rank() <<" HEMI rank"<< HEMI.rank() <<"] result:"<< H[0] <<std::endl;

//  assert( magnesium.count() == 2 );

//  auto magnesium2 = std::move(magnesium);
//  assert( magnesium2.count() == 2 );

	switch(magnesium.rank()) {
	case 0: {
		magnesium.send_n(A.data(), A.size(), 1);
	}
	case 1: {
		thrust::device_vector<T, thrust::cuda::allocator<T>> B(1000, T{});
		magnesium.receive_n(B.data(), B.size(), 0);
		assert( A == B );
	}
	}

	int i = 0;
	if(not magnesium.rank()) { i = 99; }

	magnesium.broadcast_n(&i, 1);
	assert( i == 99 );

	return 0;
}
