#include "../../../mpi3/nccl/communicator.hpp"
#include "../../../mpi3/main.hpp"

#include <thrust/system/cuda/memory.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace mpi3 = boost::mpi3;

int mpi3::main(int /*argc*/, char** /*argv*/, mpi3::communicator world) {
	assert(world.size() == 4);

	hipSetDevice(world.rank());

	auto hemi = world / 2;

//	if(world.rank() < 2) {  // this conditional is to show that it works in single subcomm

	mpi3::nccl::communicator magnesium{hemi};

	//  thust::device_vector<int64_t, thrust::cuda::universal_allocator<int64_t>> A(1000, world.rank());
	thrust::device_vector<int64_t, thrust::cuda::allocator<int64_t>> A(1000, world.rank());

	magnesium.all_reduce_n(A.data(), A.size(), A.data());
	thrust::host_vector<int64_t> H = A;

	std::cout<<"[rank"<< world.rank() <<"] result:"<< H[0] <<std::endl;
	
	switch(world.rank()) {
	       case 0: assert(H[0] == 1);
	break; case 1: assert(H[0] == 1);
	break; case 2: assert(H[0] == 5);
	break; case 3: assert(H[0] == 5);
	}

//	}
	return 0;
}
