#include "../../../mpi3/nccl/communicator.hpp"
#include "../../../mpi3/main.hpp"

#include <thrust/system/cuda/memory.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace mpi3 = boost::mpi3;

int mpi3::main(int /*argc*/, char** /*argv*/, mpi3::communicator world) {
	assert(world.size() == 4);

//	hipSetDevice(world_rank); // GPU N binds to MPI rank N

	auto hemi = world / 2;

	mpi3::nccl::communicator magnesium{hemi};

//  thust::device_vector<int64_t, thrust::cuda::universal_allocator<int64_t>> A(1000, world.rank());
	thrust::device_vector<int64_t, thrust::cuda::allocator<int64_t>> A(1000, world.rank());

	magnesium.all_reduce_n(A.data(), A.size(), A.data());
	thrust::host_vector<int64_t> H = A;

	std::cout<<"[rank"<< world.rank() <<"] result:"<< H[0] <<std::endl;

	return 0;
}
