#include "../../../mpi3/nccl/communicator.hpp"
#include "../../../mpi3/main.hpp"

#include <thrust/system/cuda/memory.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <thrust/complex.h>

namespace mpi3 = boost::mpi3;

int mpi3::main(int /*argc*/, char** /*argv*/, mpi3::communicator world) {
	assert(world.size() == 4);

//	hipSetDevice(world_rank); // GPU N binds to MPI rank N

	auto hemi = world / 2;

	mpi3::nccl::communicator magnesium{hemi};

	using T = thrust::complex<double>;  // int64_t;
//  thust::device_vector<T, thrust::cuda::universal_allocator<T>> A(1000, world.rank());
	thrust::device_vector<T, thrust::cuda::allocator<T>> A(1000, T{1.*world.rank()});

	magnesium.all_reduce_n(A.data(), A.size(), A.data());
	thrust::host_vector<T> H = A;

	std::cout<<"[rank"<< world.rank() <<"] result:"<< H[0] <<std::endl;

//	assert( magnesium.count() == 2 );

//	auto magnesium2 = std::move(magnesium);
//	assert( magnesium2.count() == 2 );

	switch(magnesium.rank()) {
	case 0: {
		magnesium.send_n(A.data(), A.size(), 1);
	}
	case 1: {
		thrust::device_vector<T, thrust::cuda::allocator<T>> B(1000, T{});
		magnesium.receive_n(B.data(), B.size(), 0);
		assert( A == B );
	}
	}

	return 0;
}
