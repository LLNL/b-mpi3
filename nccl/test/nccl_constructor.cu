#include "../../../mpi3/nccl/communicator.hpp"
#include "../../../mpi3/main.hpp"

#include "/home/correaa/prj/alf/boost/multi/include/multi/array.hpp"
#include <thrust/system/cuda/memory.h>

namespace mpi3 = boost::mpi3;
namespace multi = boost::multi;

int mpi3::main(int /*argc*/, char** /*argv*/, mpi3::communicator world) {
	assert(world.size() == 4);

//	hipSetDevice(world_rank); // GPU N binds to MPI rank N

	auto hemi = world / 2;

	mpi3::nccl::communicator magnesium{hemi};

//  multi::array<int64_t, 1, thrust::cuda::universal_allocator<int64_t>> A({1000}, world.rank());
	multi::array<int64_t, 1, thrust::cuda::allocator<int64_t>> A({1000}, world.rank());

	magnesium.all_reduce_n(A.data_elements(), A.num_elements(), A.data_elements());
	multi::array<int64_t, 1> H = A;

	std::cout<<"[rank"<< world.rank() <<"] result:"<< H[0] <<std::endl;

	return 0;
}
